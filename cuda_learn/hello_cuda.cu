
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_cuda()
{
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main()
{
    hello_cuda<<<4, 4>>>();
    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(hipError_t));
        return 1;
    }
    else {
        printf("No CUDA error\n");
    }
    hipDeviceSynchronize();
}
